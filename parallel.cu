
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <ctime>
#include <chrono>

// CUDA Kernel to convert 3D index layout to unrolled 1D layout
__global__ void getIndex(float *Un, float *Unp1, const int nx, const int ny, const int nz, const float a, const float dt, const float dx2, const float dy2, const float dz2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= 1 && i < nx - 1 && j >= 1 && j < ny - 1 && k >= 1 && k < nz - 1)
    {
        int index = i * ny * nz + j * nz + k;
        float uijk = Un[index];
        float uim1jk = Un[(i - 1) * ny * nz + j * nz + k];
        float uip1jk = Un[(i + 1) * ny * nz + j * nz + k];
        float uijm1k = Un[i * ny * nz + (j - 1) * nz + k];
        float uijp1k = Un[i * ny * nz + (j + 1) * nz + k];
        float uijkm1 = Un[i * ny * nz + j * nz + (k - 1)];
        float uijkp1 = Un[i * ny * nz + j * nz + (k + 1)];

        Unp1[index] = uijk + a * dt * ((uim1jk - 2.0 * uijk + uip1jk) / dx2 + (uijm1k - 2.0 * uijk + uijp1k) / dy2 + (uijkm1 - 2.0 * uijk + uijkp1) / dz2);
    }
}

int main()
{
    // Width, Height and Depth of the area
    const int nx = 100;   
    const int ny = 100;   
    const int nz = 100; 

    // Thermal Conductivity constant
    const float a = 0.5;     

    // Horizontal, Vertical and Depth grid spacing
    const float dx = 0.01;    
    const float dy = 0.01;   
    const float dz = 0.01;  

    const float dx2 = dx * dx;
    const float dy2 = dy * dy;
    const float dz2 = dz * dz;

    // Number of time steps
    const int numSteps = 10000; 

    // Largest stable time step
    const float dt = dx2 * dy2 * dz2 / (2.0 * a * (dx2 * dy2 + dx2 * dz2 + dy2 * dz2)); 

    int numElements = nx * ny * nz;
    
    // Allocate two sets of data for current and next timesteps on GPU
    float *d_Un, *d_Unp1;
    hipMalloc(&d_Un, numElements * sizeof(float));
    hipMalloc(&d_Unp1, numElements * sizeof(float));

    // Initializing the data with a pattern of sphere of radius of 1/6 of the width
    float *Un = new float[numElements];
    float radius2 = (nx / 6.0) * (nx / 6.0);
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            for (int k = 0; k < nz; k++)
            {
                int index = i * ny * nz + j * nz + k;
                // Distance of point from the origin
                float ds2 = (i - nx / 2) * (i - nx / 2) + (j - ny / 2) * (j - ny / 2) + (k - nz / 2) * (k - nz / 2);
                if (ds2 < radius2)
                {
                    Un[index] = 65.0;
                }
                else
                {
                    Un[index] = 5.0;
                }
            }
        }
    }
    hipMemcpy(d_Un, Un, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Unp1, Un, numElements * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // Time Start
    auto start = std::chrono::system_clock::now();

    // Main loop
    for (int n = 0; n <= numSteps; n++)
    {
        getIndex<<<numBlocks, threadsPerBlock>>>(d_Un, d_Unp1, nx, ny, nz, a, dt, dx2, dy2, dz2);
        hipDeviceSynchronize();
        std::swap(d_Un, d_Unp1);
    }

    // Time End
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start;
    std::cout << "Elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

    // Release the memory
    delete[] Un;

    hipFree(d_Un);
    hipFree(d_Unp1);

    return 0;
}
